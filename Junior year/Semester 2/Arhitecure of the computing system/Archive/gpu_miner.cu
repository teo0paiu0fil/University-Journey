#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(uint64_t *nonce, size_t size, BYTE *block_content, BYTE *block_hash, BYTE *diff) {
	if (*nonce) return;
	
	uint64_t nonce_try = threadIdx.x + blockDim.x * blockIdx.x;
	
	// make the nonce a string 
	char nonce_string[NONCE_SIZE];
	intToString(nonce_try, nonce_string);

	// concat the nonce to the block content
	char block_content_aux[BLOCK_SIZE];
	char block_hash_aux[SHA256_HASH_SIZE];
	
	d_strcpy(block_content_aux, (const char *)block_content);
	d_strcpy(block_content_aux + size, nonce_string);
	
	// apply the hash algo
	apply_sha256((BYTE *)block_content_aux, d_strlen(block_content_aux), (BYTE *)block_hash_aux, 1);

	// check results
	if (compare_hashes((BYTE *)block_hash_aux, diff) <= 0) {
	    	// if it is already found 
		if (*nonce) return;
		
		// else copy result
    		atomicExch((unsigned long long *)nonce, (unsigned long long)nonce_try);
		d_strcpy((char *)block_hash, block_hash_aux);
	   	
        }

}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	uint64_t *nonce_gpu;
	hipMalloc((void **) &nonce_gpu, sizeof(uint64_t));

	BYTE *block_content_gpu;
	hipMalloc((void **) &block_content_gpu, sizeof(BYTE) * BLOCK_SIZE);

	BYTE *block_hash_gpu;
	hipMalloc((void **) &block_hash_gpu, sizeof(BYTE) * SHA256_HASH_SIZE);

	BYTE *diff_gpu;	
	hipMalloc((void **) &diff_gpu, sizeof(BYTE) * SHA256_HASH_SIZE);

	if (block_content_gpu == 0 || diff_gpu == 0 ||  nonce_gpu == 0 || block_hash_gpu == 0) {
		printf("Eroare de alocare!\n");
		exit(1);
	}

	hipMemcpy(block_content_gpu, block_content, sizeof(BYTE) * BLOCK_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(diff_gpu, DIFFICULTY, sizeof(BYTE) * SHA256_HASH_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(nonce_gpu, &nonce, sizeof(uint64_t), hipMemcpyHostToDevice);

	uint64_t block_size = 128; 
	uint64_t blocks_no = MAX_NONCE / block_size;

	if (100000000 % block_size) {
		++blocks_no;
	}

	hipEvent_t start, stop;
    startTiming(&start, &stop);

	findNonce<<<blocks_no, block_size>>>(nonce_gpu, current_length, block_content_gpu, block_hash_gpu, diff_gpu);
	hipDeviceSynchronize();

	float seconds = stopTiming(&start, &stop);

	hipMemcpy(&nonce, nonce_gpu, sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(block_hash, block_hash_gpu, sizeof(BYTE) * SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
	
	printResult(block_hash, nonce, seconds);

	return 0;
}
